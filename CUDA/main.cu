#include "hip/hip_runtime.h"
#include "common.h"

#include <hiprand/hiprand_kernel.h>

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%f ",  arr[i]);
  }
  printf("\n");
}

__global__ void device_array_fill(float *arr, int length, unsigned long long seed, int array_fill_type)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(seed, i, 0, &state);

    if (i < length)
    {
        if (array_fill_type == 0)
        {
            arr[i] = hiprand_uniform(&state);
        }
        else if (array_fill_type == 1)
        {
            arr[i] = (float)i;
        }
        else if (array_fill_type == 2)
        {
            arr[i] = (float)(length - i);
        }
    }
}

__global__ void device_sort_check(const float *arr, int length, int *sorted) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < length - 1) 
    {
        if (arr[i] > arr[i + 1]) 
        {
            atomicExch(sorted, 0);
        }
    }
}

int main(int argc, char *argv[])
{
    int NUM_VALS = atoi(argv[1]);
    int THREADS = atoi(argv[2]);
    int array_fill_type = atoi(argv[3]);
    int sort_alg = atoi(argv[4]);

    int BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);

    // Allocate array on device
    float *values = (float*) malloc( NUM_VALS * sizeof(float));
    float *dev_values;
    size_t size = NUM_VALS * sizeof(float);
    hipMalloc((void**) &dev_values, size);

    // Allocate sorted flag on device
    int sorted = 1;
    int* dev_sorted; // Device pointer
    hipMalloc((void**)&dev_sorted, sizeof(int));

    //MEM COPY FROM HOST TO DEVICE

    // Start host to device event
    CALI_MARK_BEGIN(cudaMemcpy_host_to_device);

    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_sorted, &sorted, sizeof(int), hipMemcpyHostToDevice);

    // End  host to device Event
    hipDeviceSynchronize();
    CALI_MARK_END(cudaMemcpy_host_to_device);

    // Start Array Fill Event
    CALI_MARK_BEGIN(array_fill_name);
    srand(time(NULL));
    device_array_fill<<<BLOCKS, THREADS>>>(dev_values, NUM_VALS, (unsigned long long)rand(), array_fill_type);

    // End Array Fill Event
    hipDeviceSynchronize();
    CALI_MARK_END(array_fill_name);

    if (sort_alg == 0)
    {
        bitonic_sort(values, dev_values, NUM_VALS, THREADS, BLOCKS);
    } else if(sort_alg==1) {
        quicksort(values, dev_values, NUM_VALS, THREADS, BLOCKS);
    }

    device_sort_check<<<BLOCKS, THREADS>>>(dev_values, NUM_VALS, dev_sorted);

    //MEM COPY FROM DEVICE TO HOST

    // Start device to host event
    CALI_MARK_BEGIN(cudaMemcpy_device_to_host);

    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipMemcpy(&sorted, dev_sorted, sizeof(int), hipMemcpyDeviceToHost);

    // End device to host Event
    hipDeviceSynchronize();
    CALI_MARK_END(cudaMemcpy_device_to_host);

    hipFree(dev_values);

    if (sorted)
    {
        printf("Array is sorted\n");
    }
    else
    {
        printf("Array is not sorted\n");
    }

    array_print(values, NUM_VALS);
}